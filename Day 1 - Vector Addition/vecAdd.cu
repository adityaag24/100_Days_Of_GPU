#include <hip/hip_runtime.h>
#include <stdio.h>
#define N   1024
#define TPB 64
#define BLOCKS (N + TPB - 1) / TPB
__global__ void vecAdd(int *a, int *b, int *c){
    int bNum = blockIdx.z  * (gridDim.y  * gridDim.x)  + blockIdx.y  * gridDim.x  + blockIdx.x;
    int tNum = threadIdx.z * (blockDim.y * blockDim.x) + threadIdx.y * blockDim.x + threadIdx.x;
    int idx  = bNum * (blockDim.z * blockDim.y * blockDim.x) + tNum;
    if (idx < N) {
        c[idx] = a[idx] + b[idx];
    }
}
int main() {
    int *h_arr = (int *)malloc(N * sizeof(int));
    int *d_arr;
    hipError_t err;
    err = hipMalloc((void **)&d_arr, N * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory: %s\n", hipGetErrorString(err));
        return -1;
    }
    for (int i = 0; i < N; i++) {
        h_arr[i] = i;
    }
    err = hipMemcpy(d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to device: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        return -1;
    }
    int *d_out;
    err = hipMalloc((void **)&d_out, N * sizeof(int));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating output device memory: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        return -1;
    }
    vecAdd<<<BLOCKS, TPB>>>(d_arr, d_arr, d_out);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error launching kernel: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        hipFree(d_out);
        return -1;
    }
    err = hipMemcpy(h_arr, d_out, N * sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to host: %s\n", hipGetErrorString(err));
        hipFree(d_arr);
        hipFree(d_out);
        return -1;
    }
    for (int i = 0; i < N; i++) {
        if (h_arr[i] != 2 * i) {
            fprintf(stderr, "Error: h_arr[%d] = %d, expected %d\n", i, h_arr[i], 2 * i);
            free(h_arr);
            hipFree(d_arr);
            hipFree(d_out);
            return -1;
        }
    }
    printf("All values are correct!\n");
}