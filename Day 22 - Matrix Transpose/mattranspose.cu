#include <hip/hip_runtime.h>
#include <stdio.h>
#define TPB 32
#define NR 4096
#define NC 4096
__global__ void transpose(int *mat, int *out){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < NR && col < NC) {
        out[col * NR + row] = mat[row * NC + col];
    }
}
int main() {
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time_ms = 0.0f;
    int *h_mat, *h_out;
    int *d_mat, *d_out;
    h_mat = (int *)malloc(NR * NC * sizeof(int));
    h_out = (int *)malloc(NC * NR * sizeof(int));
    hipMalloc((void **)&d_mat, NR * NC * sizeof(int));
    hipMalloc((void **)&d_out, NC * NR * sizeof(int));
    for (int i = 0; i < NR; i++){
        for (int j = 0; j < NC; j++) {
            h_mat[i * NC + j] = rand() % 10;
        }
    }
    hipMemcpy(d_mat, h_mat, NR * NC * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_out, h_out, NC * NR * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    dim3 grid((NC + TPB - 1) / TPB, (NR + TPB - 1) / TPB, 1);
    dim3 block(TPB, TPB, 1);
    transpose<<<grid, block>>>(d_mat, d_out);
    hipEventRecord(stop,  0);
    hipEventSynchronize(stop);
    hipMemcpy(h_mat, d_out, NC * NR * sizeof(int), hipMemcpyDeviceToHost);
    hipEventElapsedTime(&time_ms, start, stop);
    printf("Time Taken = %0.2f ms\n", time_ms);
    hipFree(d_mat);
    hipFree(d_out);
    free(h_mat);
    free(h_out);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}