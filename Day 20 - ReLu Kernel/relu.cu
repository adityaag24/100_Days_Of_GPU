#include <stdio.h>
#include <hip/hip_runtime.h>
#define N   (1e8)
#define TPB 32
// Error checking macro
#define CHECK_CUDA_ERROR(call) \
{ \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA Error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
        exit(1); \
    } \
}
__global__ void reluKernel(float *input) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < N) {
        input[tid] = (input[tid] > 0)? input[tid]: 0;
    }
}
int main() {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float *h_input = (float *)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++){
        h_input[i] = (float)(rand()) / (float)(RAND_MAX);
    }
    float *d_input = NULL;
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_input, N * sizeof(float)));
    dim3 grid ((N + TPB - 1)/TPB, 1, 1);
    dim3 block(TPB, 1, 1);
    CHECK_CUDA_ERROR(hipEventRecord(start));
    CHECK_CUDA_ERROR(hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice));
    reluKernel<<<grid, block>>>(d_input);
    CHECK_CUDA_ERROR(hipMemcpy(h_input, d_input, N * sizeof(float), hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipEventRecord(stop));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    float timeTaken;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&timeTaken, start, stop));
    int nElements = N;
    printf("Time Taken for %d elements = %.2f ms\n", nElements, timeTaken);
    printf("Bandwidth = %.2f MB/s\n", (2 * N * sizeof(float)) / (timeTaken * (1<<20)));
    free(h_input);
    hipFree(d_input);
}