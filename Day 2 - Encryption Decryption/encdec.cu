#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void encryptKernel(char *message, int length) {
    int bNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
    int tNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
    int tid  = bNum * (blockDim.x * blockDim.y * blockDim.z) + tNum;
    if (tid < length){
        if (message[tid] == 'z'){
            message[tid] = 'a';
        } else {
            message[tid] = message[tid] + 1;
        }
    }
}
__global__ void decryptKernel(char *message, int length) {
    int bNum = blockIdx.z * (gridDim.x * gridDim.y) + blockIdx.y * gridDim.x + blockIdx.x;
    int tNum = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
    int tid = bNum * (blockDim.x * blockDim.y * blockDim.z) + tNum;
    if (tid < length) {
        if (message[tid] == 'a') {
            message[tid] = 'z';
        } else {
            message[tid] = (message[tid] - 1);
        }
    }
}

int main() {
    char message[] = "hundreddaysofgpu";
    int length = sizeof(message) - 1;
    char *d_message;
    hipError_t err;
    hipEvent_t start, stop;
    float elapsedTime;
    err = hipEventCreate(&start);
    if (err != hipSuccess) {
        fprintf(stderr, "Error creating start event: %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipEventCreate(&stop);
    if (err != hipSuccess) {
        fprintf(stderr, "Error creating stop event: %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipMalloc((void**)&d_message, length * sizeof(char));
    if (err != hipSuccess) {
        fprintf(stderr, "Error allocating device memory: %s\n", hipGetErrorString(err));
        return 1;
    }
    hipEventRecord(start, 0);
    err = hipMemcpy(d_message, message, length * sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to device: %s\n", hipGetErrorString(err));
        return 1;
    }
    dim3 block(8, 8, 1);
    dim3 grid(1, 1, 1);
    encryptKernel<<<grid, block>>>(d_message, length);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error launching kernel: %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipMemcpy(message, d_message, length * sizeof(char), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to host: %s\n", hipGetErrorString(err));
        return 1;
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Encrypted message: %s\n", message);
    printf("Time taken for encryption: %.2f ms\n", elapsedTime);
    printf("Bandwidth Utilization = %.2fKB/s\n", ((float)length * sizeof(char) * 2) / (elapsedTime * 1e3));
    hipEventRecord(start, 0);
    err = hipMemcpy(d_message, message, length * sizeof(char), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to device: %s\n", hipGetErrorString(err));
        return 1;
    }
    decryptKernel<<<grid, block>>>(d_message, length);
    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Error launching kernel: %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipMemcpy(message, d_message, length * sizeof(char), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying data to host: %s\n", hipGetErrorString(err));
        return 1;
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Decrypted message: %s\n", message);
    printf("Time taken for decryption: %.2f ms\n", elapsedTime);
    printf("Bandwidth Utilization = %.2fKB/s\n", ((float)length * sizeof(char) * 2) / (elapsedTime * 1e3));
    err = hipFree(d_message);
    if (err != hipSuccess) {
        fprintf(stderr, "Error freeing device memory: %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipEventDestroy(start);
    if (err != hipSuccess) {
        fprintf(stderr, "Error destroying start event: %s\n", hipGetErrorString(err));
        return 1;
    }
    err = hipEventDestroy(stop);
    if (err != hipSuccess) {
        fprintf(stderr, "Error destroying stop event: %s\n", hipGetErrorString(err));
        return 1;
    }
    return 0;
}