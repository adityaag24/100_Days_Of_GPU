#include <hip/hip_runtime.h>
#include <stdio.h>
#define TPB 32
#define r_A 4096
#define c_A 4096
#define r_B 4096
#define c_B 4096
__global__ void matmul(int *a, int *b, int *c){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (row < r_A && col < c_B) {
        for (int i = 0; i < r_B; i++) {
            sum += (a[row * r_B + i] * b[i * c_B + col]);
        }
        c[row * c_B + col] = sum; 
    }
}
__host__ void printMatrix(int *a, int r, int c){
    for(int i = 0; i < r; i++) {
        for (int j = 0; j < c; j++){
            printf("%d ", a[i*c + j]);
        }
        printf("\n");
    }
}
__host__ void matmulHost(int *a, int *b, int *c){
    for (int i = 0; i < r_A; i++){
        for (int j = 0; j < c_B; j++){
            int sum = 0;
            for (int l = 0; l < c_A; l++){
                sum += (a[i * c_A + l]*b[l * c_B + j]);
            }
            c[i * c_B + j] = sum;
        }
    }
}
int main() {
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float time_ms = 0.0f;
    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;
    h_a = (int *)malloc(r_A * c_A * sizeof(int));
    h_b = (int *)malloc(r_B * c_B * sizeof(int));
    h_c = (int *)malloc(r_A * c_B * sizeof(int));
    hipMalloc((void **)&d_a, r_A * c_A * sizeof(int));
    hipMalloc((void **)&d_b, r_B * c_B * sizeof(int));
    hipMalloc((void **)&d_c, r_A * c_B * sizeof(int));
    for (int i = 0; i < r_A; i++){
        for (int j = 0; j < c_A; j++) {
            h_a[i * c_B + j] = rand() % 10;
        }
    }
    for (int i = 0; i < r_B; i++) {
        for (int j = 0; j < c_B; j++) {
            h_b[i * c_B + j] = rand() % 10;
        }
    }
    hipMemcpy(d_a, h_a, r_A * c_A * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, r_B * c_B * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    dim3 grid((c_B + TPB - 1) / TPB, (r_A + TPB - 1) / TPB, 1);
    dim3 block(TPB, TPB, 1);
    matmul<<<grid, block>>>(d_a, d_b, d_c);
    hipEventRecord(stop,  0);
    hipEventSynchronize(stop);
    hipMemcpy(h_c, d_c, r_A * c_B * sizeof(int), hipMemcpyDeviceToHost);
    hipEventElapsedTime(&time_ms, start, stop);
    // printMatrix(h_a, r_A, c_A);
    // printMatrix(h_b, r_B, c_B);
    // printMatrix(h_c, r_A, c_B);
    printf("Time Taken = %0.2f ms\n", time_ms);
    int *v_c = (int *)malloc(r_A * c_B * sizeof(int));
    matmulHost(h_a,h_b,v_c);
    for (int i = 0; i < r_A; i++){
        for (int j = 0; j < c_B; j++){
            if (v_c[i * c_B + j] != h_c[i * c_B + j]){
                printf("Output Differs for (%d,%d) = %d vs %d\n", i, j, v_c[i * c_B + j], h_c[i * c_B + j]);
                return 0;
            }
        }
    }
    printf("Output Verified\n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);
    free(v_c);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}