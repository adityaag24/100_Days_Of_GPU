#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include "csrRep.h"
#include <time.h>
#define TPB 16
static int s2i(const char *s){
    char *p; intmax_t r;
    errno = 0;
    r = strtoimax(s, &p, 10);
    if (errno != 0 || *p != '\0' || r <= 0 || r >= INT_MAX) {
        BAIL("s2i(\"%s\") -> %" PRIdMAX ", errno => %s\n", s, r, ERRSTR);
    }
    return (int)r;
}
__host__   void printDegreeCPU(CSR h_csr) {
    for (int a = 0; a < h_csr.V; a++){
        printf("Degree of Vertex %d = %d\n", a, h_csr.N[a+1] - h_csr.N[a]);
    }
}
__global__ void printDegreeGPU(int *d_N, int *d_F, int *d_V, int *d_E) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < *d_V) {
        printf("Degree of Vertex %d = %d\n", tid, d_N[tid + 1] - d_N[tid]);
    }
}
int main(int argc, char **argv) {
    int a, b, line = 0, t = 0;
    hipEvent_t start;
    hipEvent_t stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));
    float wt;
    FILE *fp;
    CSR csr;
    if (argc != 4)
        BAIL("Usage: %s V E edgeListFile\n", argv[0]);
    csr.V = s2i(argv[1]);
    csr.E = s2i(argv[2]);
    if ((fp = fopen(argv[3], "r")) == NULL) 
        BAIL("fopen(%s) = %s", argv[3], ERRSTR);
    //Allocate Memory for the CSR Structure
    CALI(csr.N, 2 + (size_t)csr.V, sizeof(*csr.N));
    CALI(csr.F, 0 + (size_t)csr.E, sizeof(*csr.F));
    CALF(csr.W, 0 + (size_t)csr.E, sizeof(*csr.W));
    while (fscanf(fp,"%d %d %f", &a, &b, &wt) == 3) {
        line++;
        if (a > csr.V || b > csr.V)
            BAIL("%d: Bad Vertex Id: %d,%d\n", line, a, b);
        if (a == b){
            fprintf(stderr, "Line %d Same Vertex = %d\n",line, a);
        }
        csr.N[a]++;
    }
    if (!feof(fp))
        BAIL("Parse Error after %d lines: %s\n", line, ERRSTR);
    if (line != csr.E) 
        BAIL("Number of Edges (%d) is not same as the lines in the input file (%d)", csr.E, line);
    for (a = 0; a <= csr.V; a++) {
        t        += csr.N[a];
        csr.N[a]  = t;
    }
    assert(csr.N[csr.V] == csr.E);
    rewind(fp);
    while (fscanf(fp,"%d %d %f", &a, &b, &wt) == 3) {
        int idx = --csr.N[a];
        csr.F[idx] = b;
        csr.W[idx] = wt;
    }
    if (fclose(fp) != 0)
        BAIL("fclose():%s\n", ERRSTR);
    int *d_N, *d_F, *d_V, *d_E;
    float *d_W;
    clock_t hostT;
    hostT = clock();
    printDegreeCPU(csr);
    hostT = clock() - hostT;
    double hostTime = ( ((double)hostT)/CLOCKS_PER_SEC ) * 1e3;
    CHECK_CUDA_ERROR(hipEventRecord(start, 0));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_V, sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_E, sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_N, (csr.V + 1) * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_F, (csr.E + 0) * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_W, (csr.E + 0) * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemcpy(d_V, &(csr.V), sizeof(csr.V), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_E, &(csr.E), sizeof(csr.E), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_N, (csr.N), (csr.V + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_F, (csr.F), (csr.E + 0) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_W, (csr.W), (csr.E + 0) * sizeof(float), hipMemcpyHostToDevice));
    int nBlocks = (csr.V + TPB - 1) / TPB;
    hipError_t err;
    printDegreeGPU<<<nBlocks, TPB>>>(d_N, d_F, d_V, d_E);
    CHECK_CUDA_ERROR(hipEventRecord(stop, 0));
    hipEventSynchronize(stop);
    err = hipGetLastError();
    if (err!=hipSuccess) {
        fprintf(stderr, "%s\n", hipGetErrorString(err));
    } else {
        float timeTaken;
        hipEventElapsedTime(&timeTaken, start, stop);
        printf("Time Taken by CPU = %0.2lf ms\n", hostTime);
        printf("Time Taken by GPU = %0.2f ms\n", timeTaken);
        printf("Speedup Obtained  = %d\n", (int) (hostTime/timeTaken));
    }
}