#include <stdio.h>
#include <hip/hip_runtime.h>
#define THREADS_PER_BLOCK 128
#define N 8192
__global__ void reduce(int *g_iData, int *g_oData) {
    extern __shared__ int sdata[];
    // Load shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    sdata[tid] = (i < N)?g_iData[i] + g_iData[i + blockDim.x] : 0;
    __syncthreads();
 
    // Do reduction in shared memory
    for (unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    // Write result for this block to global memory
    if (tid == 0) {
        g_oData[blockIdx.x] = sdata[0];
    }
}
void runReduction(int *hData, int h_result) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int *g_iData = NULL;
    int *g_oData = NULL;
    int d_result = 0;
    float timeTaken = 0.0f;
    hipMalloc(&g_iData, N * sizeof(int));
    hipMalloc(&g_oData, N * sizeof(int));
    hipMemcpy(g_iData, hData, N * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(start, 0);
    int blocks    = (N + THREADS_PER_BLOCK - 1)/ THREADS_PER_BLOCK / 2;
    int oldBlocks = blocks;
    while (blocks > 0) {
        reduce<<<blocks, THREADS_PER_BLOCK, THREADS_PER_BLOCK * sizeof(int)>>>(g_iData,g_oData);
        hipMemcpy(g_iData, g_oData, N * sizeof(int), hipMemcpyDeviceToDevice);
        oldBlocks = blocks;
        blocks    = blocks / THREADS_PER_BLOCK / 2;
    }
    if (blocks == 0 && oldBlocks != 1) {
        reduce<<<1, oldBlocks/2, oldBlocks  * sizeof(int)>>>(g_iData, g_oData);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeTaken, start, stop);
    hipMemcpy(&d_result, &g_oData[0], sizeof(int), hipMemcpyDeviceToHost);
    if (d_result == h_result) {
        printf("Results Matched.\n");
        printf("Time Taken: %.2f ms\n", timeTaken);
        printf("Bandwidth: %.2f MB/s\n", (N * 2 * sizeof(int)) / (timeTaken * 1e6));
    } else {
        printf("Results Mismatched.\n");
        printf("Expected: %d, Got: %d\n", h_result, d_result);
    }
    hipFree(g_iData);
    hipFree(g_oData);
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
int main() {
    int hData[N];
    int hResult = 0;
    for (int i = 0; i < N; i++){
        hData[i] = rand() % 10;
        hResult += hData[i];
    }
    runReduction(hData, hResult);
}